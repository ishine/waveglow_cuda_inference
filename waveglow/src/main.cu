#include "hip/hip_runtime.h"
//# define NDEBUG // switch off all the assert calls. 
//#undef NDEBUG


#include<WN.hpp>
#include<upsample.hpp>
#include<hparams.hpp>
#include<logger.hpp>

#include<conv.hpp>
#include<data_types.hpp>
#include<cnpy.hpp>
#include<string>
#include<vector>
#include <chrono>
#include <unistd.h>

using namespace livai::tts;
using namespace livai::tts::common;

void testupsampler(hipdnnHandle_t& cudnn)
	{
		using namespace livai::tts::waveglow;
		using namespace std;
		using namespace livai::tts::common ;

		upsample upsample;

		std::cout<<"test upsample code is running"<<"\n";
		auto input_m = cnpy::npy_load("/shared1/saurabh.m/waveglow/input_mel.npy");

		gpu_float_array input_mel, upsampled_mel, d_workspace;

		input_mel.init(input_m.shape);
		upsampled_mel.init(640, input_m.shape[2]*32);
		d_workspace.init(1000000,1);

		hipMemcpy(input_mel.ptr, input_m.data<float_t>(), input_mel.size()*sizeof(float_t), hipMemcpyHostToDevice);
		upsample.set(cudnn, input_mel.shape[2]);

		auto start = chrono::steady_clock::now();

		// upsample(cudnn, input_mel, upsampled_mel);
		
		hipDeviceSynchronize();
		auto end = chrono::steady_clock::now();

		log_d("final mel", upsampled_mel.log("gen_upsamplee_mel.npy"));

		std::cout << "Elapsed time in milliseconds : " 
			<< chrono::duration_cast<chrono::milliseconds>(end - start).count()
			<< " ms" << std::endl;
		}

void testWN(hipdnnHandle_t& cudnn)
	{
		using namespace livai::tts::waveglow;
		using namespace std;
		using namespace livai::tts::common;

		WN wavenet;
		upsample upsample;
		size_t max_length = hparams::max_length;
		wavenet.set(cudnn, max_length);
		upsample.set(cudnn, max_length);

		std::cout<<"test waveglow code is running"<<"\n";
		auto input_m = cnpy::npy_load("/shared1/saurabh.m/waveglow/input_mel.npy");

		gpu_float_array input_mel, audio, d_workspace, upsampled_mel;
		d_workspace.init(100000000,1);
		input_mel.init(input_m.shape);
		audio.init(input_m.shape[2]*256,1);
		upsampled_mel.init(640, input_m.shape[2]*32);

		
		hipMemcpy(input_mel.ptr, input_m.data<float_t>(), input_mel.size()*sizeof(float_t), hipMemcpyHostToDevice);
		
		hipDeviceSynchronize();
		auto start = chrono::steady_clock::now(), up_end = start;
		int test_count=1;
		while(test_count>0)
		{
			upsample(cudnn, input_mel, upsampled_mel, d_workspace);
			hipDeviceSynchronize();
			up_end = chrono::steady_clock::now();

			wavenet(cudnn, upsampled_mel, audio, d_workspace);
			test_count--;
		}

		hipDeviceSynchronize();
		auto end = chrono::steady_clock::now();

		log_d("Gen_out audio in waveglow", audio.log("gen_out.npy"));

		std::cout << "Elapsed time in milliseconds : " 
			<< chrono::duration_cast<chrono::milliseconds>(end - start).count()
			<< " ms" << std::endl;

		std::cout << "Time elapsed time in upsampler in milliseconds : " 
			<< chrono::duration_cast<chrono::milliseconds>(up_end - start).count()
			<< " ms" << std::endl;

		}


int main()
{
// create a cuda handle
	hipdnnHandle_t cudnn;
	(hipSetDevice(3));
	checkCUDNN(hipdnnCreate(&cudnn));
	// testupsampler(cudnn);
	testWN(cudnn);
	hipdnnDestroy(cudnn);
}


