#include "hip/hip_runtime.h"
#include <WN.hpp>
#include <hparams.hpp>
#include <data_types.hpp>

#include<hipblas.h>
#include<iostream>
#include<vector>
#include<logger.hpp>
#include<utils.hpp>

using namespace livai::tts::waveglow;
using namespace livai::tts::common;

__forceinline__ __device__ float sigmoidf(float in) {
   return 1.f / (1.f + expf(-in));  
}


__global__ void fused_add_tanh_sigm_mul(size_t sz, float_t* f2, float* f3, float_t* dest)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;
    
        if(index < sz)
        {
            dest[index] = tanhf(f2[index]+f3[index])* sigmoidf(f2[index+sz] + f3[index+sz]);
        }
}

__global__ void affine_transform(size_t sz, float_t* audio, float_t* end_out, size_t stride)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;
    
        if(index < sz)
        {
            audio[index+stride] = (audio[index+stride]-end_out[index])/expf(end_out[index+stride]);
        }
}


__global__ void skip_res_add(size_t sz, float_t* f5, float* f1, float_t* f6, size_t stride)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;
    
        if(index < sz)
        {
            f6[index] += f5[index+stride];
            f1[index] += f5[index]; 
        }
}

__global__ void skip_add(size_t sz, float_t* f1, float* f6)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;
    
        if(index < sz)
        {
            f6[index] += f1[index];
        }
}


__global__ void copy_kernel(size_t sz, float_t* src, float_t* dest)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(index < sz)
    {
        dest[index]=src[index];
    }
}


__global__ void transpose_kernel(size_t sz, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    size_t index = blockIdx.x*blockDim.x + threadIdx.x;
    size_t i = index/ld_src, j= index%ld_src;
    size_t dest_index = j*ld_dest + i;

    if(index < sz)
    {
        dest[dest_index] = src[index];
    }
}

__global__ void concat_z(size_t sz, float_t* src, float_t* dest, float_t* z, size_t stride)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(index < sz)
    {
        if(index>=stride)
        {
            dest[index]=src[index-stride];
        }
        else
        {
            dest[index]=z[index];
        }
    }
}


void WN::set(hipdnnHandle_t& cudnn, size_t max_audio_len)
{
    input_len = max_audio_len; 
    
    n_channels = hparams::n_channels;
    n_flows = hparams::n_flows;
    n_layers = hparams::n_layers;
    n_groups = hparams::n_groups;
    n_rem_channels = hparams::n_rem_channels;

    n_threads = 512;

    for (int k=0; k<n_flows; k++)
    {   
        std::string kernel_fname = get_param_name(hparams::start_conv_weight, k);
        std::string bias_fname = get_param_name(hparams::start_conv_bias, k);   
        auto kernel_weight = cnpy::npy_load(kernel_fname); 
        auto bias_weight = cnpy::npy_load(bias_fname);

        size_t kernel_width = kernel_weight.shape[2];
        size_t in_channel_size = kernel_weight.shape[1];
        size_t out_channel_size = kernel_weight.shape[0];

        start_conv[k].init(cudnn, kernel_weight, bias_weight, 1, input_len, in_channel_size,
            1, input_len, out_channel_size, 1, kernel_width);
    }
    
    for (int k=0; k<n_flows; k++)
    {   
        size_t dilation = 1;

            for(int i=0; i<n_layers; i++)
            {
                std::string kernel_fname = get_res_name(hparams::in_conv_weight, k, i);
                std::string bias_fname = get_res_name(hparams::in_conv_bias, k, i); 
                auto kernel_weight = cnpy::npy_load(kernel_fname); 
                auto bias_weight = cnpy::npy_load(bias_fname);

                size_t kernel_width = kernel_weight.shape[2];
                size_t in_channel_size = kernel_weight.shape[1];
                size_t out_channel_size = kernel_weight.shape[0];

                in_conv[k][i].init(cudnn, kernel_weight, bias_weight, 1, input_len, in_channel_size,
                    1, input_len, out_channel_size, 1, kernel_width, 1, dilation);

                
                kernel_fname = get_res_name(hparams::cond_conv_weight, k, i);
                bias_fname = get_res_name(hparams::cond_conv_bias, k, i);   
                kernel_weight = cnpy::npy_load(kernel_fname); 
                bias_weight = cnpy::npy_load(bias_fname);

                kernel_width = kernel_weight.shape[2];
                in_channel_size = kernel_weight.shape[1];
                out_channel_size = kernel_weight.shape[0];

                cond_conv[k][i].init(cudnn, kernel_weight, bias_weight, 1, input_len, in_channel_size,
                    1, input_len, out_channel_size, 1, kernel_width);

                kernel_fname = get_res_name(hparams::res_skip_conv_weight, k, i);
                bias_fname = get_res_name(hparams::res_skip_conv_bias, k, i);   
                kernel_weight = cnpy::npy_load(kernel_fname); 
                bias_weight = cnpy::npy_load(bias_fname);

                kernel_width = kernel_weight.shape[2];
                in_channel_size = kernel_weight.shape[1];
                out_channel_size = kernel_weight.shape[0];

                res_skip_conv[k][i].init(cudnn, kernel_weight, bias_weight, 1, input_len, in_channel_size,
                    1, input_len, out_channel_size, 1, kernel_width);

                dilation*=2;

            }
    }
    for (int k=0; k<n_flows; k++)
    {   

        std::string kernel_fname = get_param_name(hparams::end_conv_weight, k);
        std::string bias_fname = get_param_name(hparams::end_conv_bias, k); 
        auto kernel_weight = cnpy::npy_load(kernel_fname); 
        auto bias_weight = cnpy::npy_load(bias_fname);

        size_t kernel_width = kernel_weight.shape[2];
        size_t in_channel_size = kernel_weight.shape[1];
        size_t out_channel_size = kernel_weight.shape[0];

        end_conv[k].init(cudnn, kernel_weight, bias_weight, 1, input_len, in_channel_size,
            1, input_len, out_channel_size, 1, kernel_width);
    
        kernel_fname = get_param_name(hparams::inv_conv_weight, k);
        bias_fname = get_param_name(hparams::end_conv_bias, k); 
        kernel_weight = cnpy::npy_load(kernel_fname); 
        bias_weight = cnpy::npy_load(bias_fname);

        kernel_width = kernel_weight.shape[2];
        in_channel_size = kernel_weight.shape[1];
        out_channel_size = kernel_weight.shape[0];

        inv_conv[k].init(cudnn, kernel_weight, bias_weight, 1, input_len, in_channel_size,
            1, input_len, out_channel_size, 1, kernel_width);
    }


    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnCreateTensorDescriptor(&out_desc);

    // std::cout<<"input length is "<<input_len<<"\n";
    {
        temp_input.init(n_groups/2, input_len);
        f1.init(n_channels, input_len);
        f2.init(2*n_channels, input_len);
        f3.init(2*n_channels, input_len);
        f4.init(n_channels, input_len);
        f6.init(n_channels, input_len);
        temp.init(n_groups, input_len);
        z.init(2, 2*input_len);
        input_t.init(n_groups,input_len);
    }

    {
        checkCURAND(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
        checkCURAND(hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull));
    }

}


void WN::operator() (hipdnnHandle_t& cudnn, gpu_float_array& mel_input, gpu_float_array& d_output, gpu_float_array& d_workspace)
{   

    size_t input_len = mel_input.shape[1];
    size_t aud_channels = n_rem_channels;
    // std::cout<<"the value is"<<input_len<<"\t"<<input_t.shape[2]<<"\t"<<mel_input.shape[1]<<"\n";

    input_t.reshape(aud_channels, input_len);
    hiprandGenerateNormal(rng, input_t.ptr, input_t.size(), 0.0f, 0.6);

    f1.reshape(n_channels, input_len);
    f2.reshape(2*n_channels, input_len);
    f3.reshape(2*n_channels, input_len);
    f4.reshape(n_channels, input_len);
    f6.reshape(n_channels, input_len);
    temp_input.reshape(aud_channels/2, input_len);
    temp.reshape(aud_channels, input_len);


    for(int k=n_flows-1; k>-1; k--)
    {
        copy_kernel <<< (temp_input.size()+n_threads-1)/n_threads, n_threads >>>(temp_input.size(), input_t.ptr, temp_input.ptr);

        hipdnnSetTensor4dDescriptor(input_desc,
                                          /*format=*/hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW,
                                          /*dataType=*/hipdnnDataType_t::HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/1,
                                          /*channels=*/aud_channels/2,
                                          /*image_height=*/1,
                                          /*image_width=*/input_len);
        
        hipdnnSetTensor4dDescriptor(out_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, n_channels, 1, input_len);
        start_conv[k](cudnn, temp_input, f1, input_desc, out_desc, d_workspace);

        f6.reset();
        for(int j=0; j<n_layers; j++)
        {
                // log_d("input", f1.log("inp_in" + std::to_string(j)+ ".npy"));

                hipdnnSetTensor4dDescriptor(input_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, n_channels, 1, input_len);
                hipdnnSetTensor4dDescriptor(out_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, 2*n_channels, 1, input_len);
                in_conv[k][j](cudnn, f1, f2, input_desc, out_desc, d_workspace);
                // log_d("in_out", f2.log("in_out" + std::to_string(j)+ ".npy"));

                hipdnnSetTensor4dDescriptor(input_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, 640, 1, input_len);
                hipdnnSetTensor4dDescriptor(out_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, 2*n_channels, 1, input_len);
                cond_conv[k][j](cudnn, mel_input, f3, input_desc, out_desc, d_workspace);
                // log_d("cond_out", f3.log("cond_out" + std::to_string(j)+ ".npy"));

                fused_add_tanh_sigm_mul <<< (f4.size()+n_threads-1)/n_threads, n_threads >>>(f4.size(), f2.ptr, f3.ptr, f4.ptr);
                // log_d("acts ", f4.log("acts_out" + std::to_string(j)+ ".npy"));

                
                if(j<7)
                {
                    hipdnnSetTensor4dDescriptor(input_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, n_channels, 1, input_len);
                    hipdnnSetTensor4dDescriptor(out_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, 2*n_channels, 1, input_len);
                    res_skip_conv[k][j](cudnn, f4, f3, input_desc, out_desc, d_workspace);
                    // log_d("res_skip_acts ", f3.log("res_skip_acts" + std::to_string(j)+ ".npy"));

                    skip_res_add <<< (f1.size()+n_threads-1)/n_threads, n_threads >>>(f1.size(), f3.ptr, f1.ptr, f6.ptr, 256*input_len);
                    // log_d("outputs ", f6.log("outputs" + std::to_string(j)+ ".npy"));
                }
                else
                {
                    hipdnnSetTensor4dDescriptor(input_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, n_channels, 1, input_len);
                    hipdnnSetTensor4dDescriptor(out_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, n_channels, 1, input_len);
                    res_skip_conv[k][j](cudnn, f4, f1, input_desc, out_desc, d_workspace);
                    // log_d("res_skip_acts ", f1.log("res_skip_acts" + std::to_string(j)+ ".npy"));

                    skip_add <<< (f1.size()+n_threads-1)/n_threads, n_threads >>>(f1.size(), f1.ptr, f6.ptr);
                    // log_d("outputs ", f6.log("outputs" + std::to_string(j)+ ".npy"));
                }

        }

        hipdnnSetTensor4dDescriptor(input_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, n_channels, 1, input_len);
        hipdnnSetTensor4dDescriptor(out_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, aud_channels, 1, input_len);
        end_conv[k](cudnn, f6, temp, input_desc, out_desc, d_workspace);
        // log_d(" end conv outputs ", temp.log("end_out.npy"));

        affine_transform <<< (temp.size()/2+n_threads-1)/n_threads, n_threads >>>(temp.size()/2, input_t.ptr, temp.ptr, temp.size()/2);
        // log_d("audio transformed", input_t.log("audio_tr.npy"));

        hipdnnSetTensor4dDescriptor(input_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, aud_channels, 1, input_len);
        hipdnnSetTensor4dDescriptor(out_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, aud_channels, 1, input_len);
        inv_conv[k](cudnn, input_t, temp, input_desc, out_desc, d_workspace, 0);

        copy_kernel<<<(input_t.size()+n_threads-1)/n_threads, n_threads>>>(input_t.size(), temp.ptr, input_t.ptr);


        if((k%4==0) && (k>0))
            {
                aud_channels +=2;

                input_t.reshape(aud_channels, input_len);
                z.reshape(2, input_len);
                hiprandGenerateNormal(rng, z.ptr, z.size(), 0.0f, 0.6);
                concat_z<<<(input_t.size()+n_threads-1)/n_threads, n_threads>>>(input_t.size(), temp.ptr, input_t.ptr, z.ptr, 2*input_len);
                
                temp_input.reshape(aud_channels/2, input_len);
                temp.reshape(aud_channels, input_len);
            }
     
        // log_d("audio transformed inv", input_t.log("audio_after_step" + std::to_string(k)+ ".npy"));

    }

    transpose_kernel<<<(d_output.size()+n_threads-1)/n_threads, n_threads>>>(d_output.size(), input_t.ptr, d_output.ptr, input_t.shape[1], input_t.shape[0]);

    // std::cout<<input_t.shape[1]<<"\t"<<input_t.shape[0]<<"\n";

}


WN::~WN()
{

}
