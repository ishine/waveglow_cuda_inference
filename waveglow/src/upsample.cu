#include "hip/hip_runtime.h"
#include <upsample.hpp>
#include <hparams.hpp>
#include <data_types.hpp>

#include<hipblas.h>
#include<iostream>
#include<vector>
#include<logger.hpp>
#include<utils.hpp>

using namespace livai::tts::waveglow;
using namespace livai::tts::common;




/* kernel to add zero padding in input to treat transposedConv1d as Conv1d, not used anymore */
__global__ void fractional_stride_nchw(size_t num_values, size_t stride, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    size_t index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index < num_values)
    {
        size_t frame_id = (index/ ld_src)*ld_dest + (stride)*(index%ld_src) ;
        dest[frame_id] = src[index];
    }
}

/* kernel to reshape srs with leading dimension (ld_src) to dest with leading dimension (ld_dest)*/
__global__ void reshape(size_t num_values, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    size_t index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index < num_values)
    {
        size_t src_index = (index/ld_dest)*ld_src+ index%ld_dest;
        dest[index] = src[src_index];
    }
}

/*kernel to apply the given series of transformation on spect
        spect = spect.unfold(2, self.n_group, self.n_group).permute(0, 2, 1, 3)
        spect = spect.contiguous().view(spect.size(0), spect.size(1), -1).permute(0, 2, 1)
*/
__global__ void transformation(size_t num_values, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    size_t index = blockIdx.x*blockDim.x + threadIdx.x;

    if(index < num_values)
    {
        size_t dest_index = (index/ld_src)*ld_src + ((index%ld_src)%8)*ld_dest+ (index%ld_src)/8;
        dest[dest_index] = src[index];
    }
}

void upsample::set(hipdnnHandle_t& cudnn, size_t max_mel_length)
/* initialize the weights and biases of the upsampling Convolution layers 
Arguments:
    --------------
    cudnn: A cudnnHandle 
        A cudnn handle used by various cudnn layers

    max_mel_length: integer to denote size of mel i.e. (len of audio)/256
        maximum mel_length that is in the dataset, so that cuda malloc can be avoided
*/

{
    size_t input_len = max_mel_length; 
    mel_dim = hparams::mel_dim;
    stride = hparams::stride; 
    n_threads = 1024;

    /* When posing conv as transpose Conv2d, we can use this conv layer
    {
        std::string kernel_fname = hparams::up_conv_weight;
        std::string bias_fname = hparams::up_conv_bias;   
        auto kernel_weight = cnpy::npy_load(kernel_fname); 
        auto bias_weight = cnpy::npy_load(bias_fname);

        size_t kernel_width = kernel_weight.shape[2];
        kernel_len = kernel_width;
        size_t in_channel_size = kernel_weight.shape[1];
        size_t out_channel_size = kernel_weight.shape[0];

        size_t input_rows = max_mel_length+(max_mel_length-1)*(stride-1);
        size_t output_rows = max_mel_length*stride+kernel_len-stride;
        up_conv.init(cudnn, kernel_weight, bias_weight, 1, input_rows, in_channel_size,
            1, output_rows, out_channel_size, 1, kernel_width);

    }
    (/)
    /* transpose conv1d initialization*/
    {
        std::string kernel_fname = hparams::up_conv_weight_orig;
        std::string bias_fname = hparams::up_conv_bias;   
        auto kernel_weight = cnpy::npy_load(kernel_fname); 
        auto bias_weight = cnpy::npy_load(bias_fname);

        size_t kernel_width = kernel_weight.shape[2];
        kernel_len = kernel_width;
        size_t in_channel_size = kernel_weight.shape[1];
        size_t out_channel_size = kernel_weight.shape[0];

        size_t input_rows = max_mel_length;
        size_t output_rows = max_mel_length*stride+kernel_len-stride;
        trans_conv.init(cudnn, kernel_weight, bias_weight, 1, input_rows, in_channel_size,
            1, output_rows, out_channel_size, 1, kernel_width);
    }
    

    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnCreateTensorDescriptor(&out_desc);

    f1.init(mel_dim, stride*input_len+1024);
    f2.init(mel_dim, stride*input_len+1024);

}


void upsample::operator() (hipdnnHandle_t& cudnn, gpu_float_array& input_mel, gpu_float_array& d_output, gpu_float_array& d_workspace)
/*
    Function to upsample the input mel
    
    Arguments:
    --------------
    cudnn: A cudnnHandle 
        A cudnn handle used by various cudnn layers

    input_mel: a float array of size [80,x]
        input-mel generated from either text-2-mel models such as tacotron, deepvoice or 
        from ground truth fft.

    d_output: A float array of size [640,x*32]
        Pointer to store values of transformed upsampled mel

    d_workspace: A float array of large size ( greater than required by any convolution)
        A chunk of memory to be used by convolution workspace, alternatively we can set size to a
         given maximum by selecting such algorithms in conv
*/

{   

    size_t input_len = input_mel.shape[2];
    size_t input_rows = input_len+(input_len-1)*(stride-1);
    size_t output_rows = input_len*stride+kernel_len-stride;

    f1.reset();
    f1.reshape(mel_dim, input_rows);
    f2.reshape(mel_dim, output_rows);
    
    hipdnnSetTensor4dDescriptor(input_desc,
                                      /*format=*/hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW,
                                      /*dataType=*/hipdnnDataType_t::HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/1,
                                      /*channels=*/mel_dim,
                                      /*image_height=*/1,
                                      /*image_width=*/input_rows);
    
    hipdnnSetTensor4dDescriptor(out_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, mel_dim, 1, output_rows);
    
    size_t num_values = input_mel.size();
    // fractional_stride_nchw<<<(num_values+n_threads-1)/n_threads, n_threads>>>(num_values, stride, input_mel.ptr, f1.ptr, input_len, input_rows);
    // up_conv(cudnn, f1, f2, input_desc, out_desc, d_workspace, 0);

    hipdnnSetTensor4dDescriptor(input_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, mel_dim, 1, input_len);
    hipdnnSetTensor4dDescriptor(out_desc, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, mel_dim, 1, output_rows);
    trans_conv(cudnn, input_mel, f2, input_desc, out_desc, d_workspace);

    size_t upsampled_dim = input_len*stride;
    f1.reshape(mel_dim, upsampled_dim);
    num_values = f1.size();
    reshape<<<(num_values+n_threads-1)/n_threads, n_threads>>>(num_values, f2.ptr, f1.ptr, output_rows, upsampled_dim);

    f2.reshape(640, upsampled_dim/8);
    transformation<<<(num_values+n_threads-1)/n_threads, n_threads>>>(num_values, f1.ptr, d_output.ptr, upsampled_dim, upsampled_dim/8);
}


upsample::~upsample()
{

}
